#include "hip/hip_runtime.h"
#include <cudaStdafx.h>
#include "hip/driver_types.h"
#include "hip/hip_runtime_api.h"
#include "dgCudaSolverKernels.h"




enum dgContactFlags
{
	m_collisionEnable__					 = 1<<0,
	m_friction0Enable__					 = 1<<1,
	m_friction1Enable__					 = 1<<2,
	m_override0Accel__					 = 1<<3,
	m_override1Accel__					 = 1<<4,
	m_overrideNormalAccel__				 = 1<<5,
	m_collisionContinueCollisionEnable__ = 1<<6,
};


#define DG_BILATERAL_CONSTRAINT				-1
#define DG_NORMAL_CONSTRAINT				-2 
#define DG_BILATERAL_FRICTION_CONSTRAINT	-3 

#define DG_PSD_DAMP_TOL						float (1.0e-3f)
#define DG_MAX_BOUND						float (1.0e15f)
#define DG_MIN_BOUND						(-DG_MAX_BOUND)
#define REST_RELATIVE_VELOCITY				float (1.0e-3f)
#define MAX_DYNAMIC_FRICTION_SPEED			float (0.3f)


//static void* m_returnValuePtr;
//static void* m_userGlobalLockPtr;
//__device__ int m_returnValue;
//__device__ int m_userGlobalLock = 0;



void dgCudaExpandTraceMessage (const char *fmt, ...)
{
	va_list v_args;
	char text[1024];

	text[0] = 0;
	va_start (v_args, fmt);     
	vsprintf(text, fmt, v_args);
	va_end (v_args);            

	OutputDebugStringA (text);
}



#define dgCudaTrace(x)				\
{									\
	dgCudaExpandTraceMessage x;		\
}																	




__device__ float DotProduct (float* const A, const float* const B)
{
	return A[0] * B[0] + A[1] * B[1] + A[2] * B[2];
}



__global__ void InertiaMatricesAndKineticForces (dgCudaBody::dgBodyInfo* const bodies);

__global__ void CalculateContactDerivative (dgCudaJointGraph::dgJointInfo* const constraintInfo,
											dgCudaBody::dgBodyInfo* const bodyArray,
											dgCudaContactMemory::dgContact* const contactMemory, 
											dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory, 
											float invTimeStep); 

__global__ void dgParallelSolverBuildJacobianRows (dgCudaJointGraph::dgJointInfo* const constraintInfo,
												   dgCudaBody::dgBodyInfo* const bodyArray,
												   dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory);

__global__ void InitInternalForces (int* const jointIndex,
									dgCudaJointGraph::dgJointInfo* const constraintInfo,
								    dgCudaBody::dgBodyInfo* const bodyArray,
								    dgCudaBody::dgBodyWorkData* const bodyInternalForces,
									dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory);
									
__global__ void CalculateRowsAcceleration (dgCudaJointGraph::dgJointInfo* const constraintInfo,
										   dgCudaBody::dgBodyInfo* const bodyArray,
										   dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory,
										   float timestep,
										   float invTimestep,
										   float firstPassCoef);
										 
										   
__global__ void CalculateJointForces (int* const jointIndex,
									  dgCudaJointGraph::dgJointInfo* const constraintInfo,
									  dgCudaBody::dgBodyInfo* const bodyArray,
									  dgCudaBody::dgBodyWorkData* const bodyForceVeloc,
									  dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory);
									
__global__ void UpdateBodiesVelocities (dgCudaBody::dgBodyInfo* const bodyArray,
										dgCudaBody::dgBodyWorkData* const bodyForceVeloc,
										float timestep);
										
								
								
__device__ void SpinLock (int* lock)
{
	if (threadIdx.x == 0) {
		while (atomicCAS(lock, 0, 1));
	}
	__syncthreads();
}
		


__device__ void SpinUnlock (int* lock)
{
	if (threadIdx.x == 0) {
		atomicExch(lock, 0); 
	}
	__syncthreads();
}
								


void* CudaMalloc(int size)
{
	void* mem;
	hipError_t error;

	error = hipMalloc ((void**)&mem, size);
	_ASSERTE (error == hipSuccess);

	return mem;
}

void CudaFree(void* ptr)
{
	hipError_t error;

	error = hipFree (ptr);
	_ASSERTE (error == hipSuccess);
}


/*
__device__ void MatrixRotateVector(float* const out, const float* const vector, const float* const matrix)
{
	int row;
	__shared__ float tmp [16]; 
	
	row = threadIdx.x >> 2;
	
	tmp[threadIdx.x] = vector[row] * matrix[threadIdx.x];
	__syncthreads();
	if (threadIdx.x < 4) {
		out[threadIdx.x] = tmp[threadIdx.x] + tmp[threadIdx.x + 4] + tmp[threadIdx.x + 8];		
	}
}

__device__ void MatrixUnrotateVector(float* const out, const float* const vector, const float* const matrix)
{
	int colum;
	__shared__ float tmp [16]; 

	colum = threadIdx.x & 3;
	tmp[threadIdx.x] = vector[colum] * matrix[threadIdx.x];
//	__syncthreads();		
	if (threadIdx.x < 4) {
		int j = threadIdx.x << 2;
		out[threadIdx.x] = tmp[j] + tmp[j+1] + tmp[j+2];		
	}
}

__device__ void MatrixMultiply(float* const out, const float* const A, const float* const B)
{
	int row;
	int colum;
	int rowScale;
	
	row = threadIdx.x >> 2;
	colum = threadIdx.x & 3;
	rowScale = row << 2;
	
	out[rowScale + colum]  = A[rowScale + 0] * B[0 + colum];
	out[rowScale + colum] += A[rowScale + 1] * B[4 + colum];
	out[rowScale + colum] += A[rowScale + 2] * B[8 + colum];
//	__syncthreads();			
}
*/

void CalculateInertiaMatricesAndBodyKineticForces (dgCudaBody& bodyArray)
{
	hipError_t error;
	if (bodyArray.m_count > 0) {	
		error = hipMemcpy (&bodyArray.m_devBody[0], &bodyArray.m_sysBody[0], bodyArray.m_count * sizeof (dgCudaBody::dgBodyInfo), hipMemcpyHostToDevice);
		_ASSERTE (error == hipSuccess);
		
		error = hipMemset (&bodyArray.m_devBodyWorkData[0], 0, bodyArray.m_count * sizeof (dgCudaBody::dgBodyWorkData));
		_ASSERTE (error == hipSuccess);	

//		int count;
//		int threads;
//		threads = 1<<(4 + DG_CLUSTER_TREAD_SIZE_BITS);
//		count = ((bodyArray.m_count - 1) >> DG_CLUSTER_TREAD_SIZE_BITS) + 1;
//		InertiaMatricesAndKineticForces <<<count, threads>>> (&bodyArray.m_devBody[0]);
		InertiaMatricesAndKineticForces <<<bodyArray.m_count, 16>>> (&bodyArray.m_devBody[0]);
	}
}


void CalculateContactDerivatives (
	dgCudaJointGraph& contraints, 
	dgCudaContactMemory& contactMemory, 
	dgCudaJacobianMemory& jacobianMemory, 
	dgCudaBody& bodyArray, 
	float timestep)	
{
	hipError_t error;
	float invTimestep;
	
	if (contraints.m_count > 0) {	
		error =	hipMemcpy (&contraints.m_devConstraintArray[0], &contraints.m_sysConstraintArray[0], contraints.m_count * sizeof (dgCudaJointGraph::dgJointInfo), hipMemcpyHostToDevice);
		_ASSERTE (error == hipSuccess);

		error =	hipMemcpy (&contactMemory.m_devContacts[0], &contactMemory.m_sysContacts[0], contactMemory.m_count * sizeof (dgCudaContactMemory::dgContact), hipMemcpyHostToDevice);
		_ASSERTE (error == hipSuccess);

		invTimestep = 1.0f / timestep;
		CalculateContactDerivative  <<<contraints.m_count, 16>>> (&contraints.m_devConstraintArray[0], 
																  &bodyArray.m_devBody[0], 
																  &contactMemory.m_devContacts[0],
																  &jacobianMemory.m_devJacobian[0],
																  invTimestep);
  }
}



void CalculateJacobianMatrix (dgCudaJointGraph& contraints, dgCudaBody& bodyArray, dgCudaJacobianMemory& jacobianMemory)
{
	if (contraints.m_count > 0) {
		dgParallelSolverBuildJacobianRows  <<<contraints.m_count, 16>>> (&contraints.m_devConstraintArray[0],
																		 &bodyArray.m_devBody[0],
																		 &jacobianMemory.m_devJacobian[0]);
	}																		 
}	


void InitInternalForces (
	int deviceVersion,
	dgCudaJointsSlack& slack, 
	dgCudaJointGraph& contraints, 
	dgCudaBody& bodyArray, 
	dgCudaJacobianMemory& jacobianMemory)
{
	if (deviceVersion >= 101) {
	
		hipError_t error;
		error =	hipMemcpy (&slack.m_slackArray[0].m_devJointSlack[0], &slack.m_slackArray[0].m_sysJointSlack[0], contraints.m_count * sizeof (int), hipMemcpyHostToDevice);
		_ASSERTE (error == hipSuccess);

		InitInternalForces  <<<contraints.m_count, 16>>> (&slack.m_slackArray[0].m_devJointSlack[0],
														  &contraints.m_devConstraintArray[0],
														  &bodyArray.m_devBody[0],
														  &bodyArray.m_devBodyWorkData[0],
														  &jacobianMemory.m_devJacobian[0]);

	
	} else {
		for (int i = 0; slack.m_slackArray[i].m_count; i ++) {
			hipError_t error;

			error =	hipMemcpy (&slack.m_slackArray[i].m_devJointSlack[0], &slack.m_slackArray[i].m_sysJointSlack[0], slack.m_slackArray[i].m_count * sizeof (int), hipMemcpyHostToDevice);
			_ASSERTE (error == hipSuccess);

			InitInternalForces  <<<slack.m_slackArray[i].m_count, 16>>> (&slack.m_slackArray[i].m_devJointSlack[0],
																		 &contraints.m_devConstraintArray[0],
																		 &bodyArray.m_devBody[0],
																		 &bodyArray.m_devBodyWorkData[0],
																		 &jacobianMemory.m_devJacobian[0]);
		}														  
	}
}	


void CalculateRowsAcceleration (
	dgCudaJointGraph& contraints, 
	dgCudaBody& bodyArray, 
	dgCudaJacobianMemory& jacobianMemory, 
	float timestep,
	float invTimestep,
	float firstPassCoef)
{
	if (contraints.m_count > 0) {
		CalculateRowsAcceleration  <<<contraints.m_count, 16>>> (&contraints.m_devConstraintArray[0],
																 &bodyArray.m_devBody[0],
																 &jacobianMemory.m_devJacobian[0],
																 timestep,
																 invTimestep,
																 firstPassCoef);
	}																 
} 


void CalculateJointForces (
	int deviceVersion,
	dgCudaJointsSlack& slack, 
	dgCudaJointGraph& contraints, 
	dgCudaBody& bodyArray, 
	dgCudaJacobianMemory& jacobianMemory)
{
	if (deviceVersion >= 101) {
		CalculateJointForces  <<<contraints.m_count, 16>>> (&slack.m_slackArray[0].m_devJointSlack[0],
											                &contraints.m_devConstraintArray[0],
															&bodyArray.m_devBody[0],
															&bodyArray.m_devBodyWorkData[0],
															&jacobianMemory.m_devJacobian[0]);
	} else {
		for (int i = 0; slack.m_slackArray[i].m_count; i ++) {
			CalculateJointForces  <<<slack.m_slackArray[i].m_count, 16>>> (&slack.m_slackArray[i].m_devJointSlack[0],
																		   &contraints.m_devConstraintArray[0],
																		   &bodyArray.m_devBody[0],
																		   &bodyArray.m_devBodyWorkData[0],
																		   &jacobianMemory.m_devJacobian[0]);
		}
	}
} 


void UpdateBodiesVelocities (dgCudaBody& bodyArray, float timestep)
{
	if (bodyArray.m_count > 0) {
		UpdateBodiesVelocities <<<bodyArray.m_count, 16>>> (&bodyArray.m_devBody[0],
															&bodyArray.m_devBodyWorkData[0],
															timestep);
	}
} 







__global__ void CalculateContactDerivative (
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaContactMemory::dgContact* const contactMemory, 
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory, 
	float invTimestep) 
{
	int block;
	int thread;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo0;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo1;
	__shared__ dgCudaContactMemory::dgContact contact;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianInfo jacobianInfo;
	__shared__ float r0[4];
	__shared__ float r1[4];
	__shared__ float veloc0[4];
	__shared__ float veloc1[4];
	__shared__ float velocError[4];
	
	block = blockIdx.x;
	thread = threadIdx.x;
	
	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];
	__syncthreads();
	

	if (jointInfo.m_contactStart >= 0) {
		int m0;
		int m1;
		int count;
		int start;
		int isMotor;
		int jacobIndex;
		int frictionIndex;
		float accel;
		float restitution;
		float penetration;
		float relVelocErr;
		float lowBound;
		float upperBound;
		float penetrationVeloc;
		float penetrationStiffness;
		
		int crossX = thread + 1;
		int crossY = thread + 2;
		if (crossX >= 3) {
			crossX -= 3;
		}
		if (crossY >= 3) {
			crossY -= 3;
		}
		if (thread > 2) {
			crossX = 3;
			crossY = 3;
		}

		count = jointInfo.m_autoPaircount;
		m0 = jointInfo.m_m0;
		m1 = jointInfo.m_m1;
		start = jointInfo.m_contactStart;
		jacobIndex = jointInfo.m_autoPairstart;
		frictionIndex = jacobIndex + count;

		bodyInfo0.m_data0[thread] = bodyArray[m0].m_data0[thread];
		bodyInfo0.m_data1[thread] = bodyArray[m0].m_data1[thread];
		bodyInfo1.m_data0[thread] = bodyArray[m1].m_data0[thread];
		bodyInfo1.m_data1[thread] = bodyArray[m1].m_data1[thread];
		__syncthreads();
		
		for (int i = 0; i < count; i ++) {
		
			contact.m_data0[thread] = contactMemory[start + i].m_data0[thread];
			contact.m_data1[thread] = contactMemory[start + i].m_data1[thread];
			__syncthreads();

			if (thread < 4) {
				r0[thread] = contact.m_point[thread] - bodyInfo0.m_globalCentreOfMass[thread];
				r1[thread] = contact.m_point[thread] - bodyInfo1.m_globalCentreOfMass[thread];
			}
			__syncthreads();

			if (thread < 4) {
				veloc0[thread] = bodyInfo0.m_veloc[thread] + bodyInfo0.m_omega[crossX] * r0[crossY] - bodyInfo0.m_omega[crossY] * r0[crossX];
				veloc1[thread] = bodyInfo1.m_veloc[thread] + bodyInfo1.m_omega[crossX] * r1[crossY] - bodyInfo1.m_omega[crossY] * r1[crossX];
				velocError[thread] = veloc1[thread] - veloc0[thread];
			}
			__syncthreads();

			if (thread < 4) {
				jacobianInfo.m_Jt.m_jacobian_IM0.m_linear[thread] = contact.m_normal[thread];
				jacobianInfo.m_Jt.m_jacobian_IM0.m_angular[thread] = r0[crossX] * contact.m_normal[crossY] - r0[crossY] * contact.m_normal[crossX];			
				
				jacobianInfo.m_Jt.m_jacobian_IM1.m_linear[thread] = -contact.m_normal[thread];
				jacobianInfo.m_Jt.m_jacobian_IM1.m_angular[thread] = contact.m_normal[crossX] * r1[crossY] - contact.m_normal[crossY] * r1[crossX];
			}
			__syncthreads();
			jacobianMemory[jacobIndex].m_Jt.m_jacobianData[thread] = jacobianInfo.m_Jt.m_jacobianData[thread];
			__syncthreads();

			if (thread == 0) {
				restitution	= contact.m_restitution;
				relVelocErr = DotProduct (velocError, contact.m_normal);
				
				penetration = (contact.m_penetration < 0.5f) ? contact.m_penetration : 0.5f;

				penetrationStiffness = 50.0f * contact.m_softness;
				penetrationVeloc = penetration * penetrationStiffness;
				if (relVelocErr > REST_RELATIVE_VELOCITY) {
					relVelocErr *= (restitution + 1.0f);
				}


				jacobianInfo.m_restitution = contact.m_restitution;
				jacobianInfo.m_penetration = penetration;
				jacobianInfo.m_penetrationStiffness = penetrationStiffness;
				jacobianInfo.m_lowerBoundFrictionCoefficent = 0.0f;
				jacobianInfo.m_upperBoundFrictionCoefficent = DG_MAX_BOUND;
				jacobianInfo.m_normalForceIndex = DG_NORMAL_CONSTRAINT;
				jacobianInfo.m_force = contact.m_normal_Force;
				jacobianInfo.m_jointFeebackForcePtr = contact.m_normal_ForcePtr;
				jacobianInfo.m_diagDamp = 1.0f;
				jacobianInfo.m_accelIsMotor = 0;
				accel = relVelocErr + penetrationVeloc;
				accel = ((accel > -4.0f) ? accel : -4.0f) * invTimestep;
				if (contact.m_flags & m_overrideNormalAccel__) {
					accel += contact.m_normal_Force;
				}
				jacobianInfo.m_coordenateAccel = accel;
			}
			__syncthreads();
			jacobianMemory[jacobIndex].m_data0[thread] = jacobianInfo.m_data0[thread];
			__syncthreads();
			
			// first dir friction force
			if (contact.m_flags & m_friction0Enable__) {
			
				if (thread < 4) {
					jacobianInfo.m_Jt.m_jacobian_IM0.m_linear[thread] = contact.m_dir0[thread];
					jacobianInfo.m_Jt.m_jacobian_IM0.m_angular[thread] = r0[crossX] * contact.m_dir0[crossY] - r0[crossY] * contact.m_dir0[crossX];			
													
					jacobianInfo.m_Jt.m_jacobian_IM1.m_linear[thread] = -contact.m_dir0[thread];
					jacobianInfo.m_Jt.m_jacobian_IM1.m_angular[thread] = contact.m_dir0[crossX] * r1[crossY] - contact.m_dir0[crossY] * r1[crossX];
				}
				__syncthreads();
				jacobianMemory[frictionIndex].m_Jt.m_jacobianData[thread] = jacobianInfo.m_Jt.m_jacobianData[thread];

				if (thread == 0) {
					relVelocErr = DotProduct (velocError, contact.m_dir0);
					jacobianInfo.m_normalForceIndex = jacobIndex;
					jacobianInfo.m_diagDamp = 1.0f;
					jacobianInfo.m_restitution = 0.0f;
					jacobianInfo.m_penetration = 0.0f;
					jacobianInfo.m_penetrationStiffness = 0.0f;
					if (contact.m_flags & m_override0Accel__) {
						accel = contact.m_dir0_Force;
						isMotor = 1;
					} else {
						accel = relVelocErr * invTimestep;
						isMotor = 0;
					}
					
					jacobianInfo.m_coordenateAccel = accel;
					jacobianInfo.m_accelIsMotor = isMotor;
					
					if (fabsf (relVelocErr) > MAX_DYNAMIC_FRICTION_SPEED) {
						lowBound = -contact.m_dynamicFriction0;
						upperBound = contact.m_dynamicFriction0;
					} else {
						lowBound = -contact.m_staticFriction0;
						upperBound = contact.m_staticFriction0;
					}
					
					jacobianInfo.m_lowerBoundFrictionCoefficent = lowBound;
					jacobianInfo.m_upperBoundFrictionCoefficent = upperBound;
					jacobianInfo.m_force = contact.m_dir0_Force;
					jacobianInfo.m_jointFeebackForcePtr = contact.m_dir0_ForcePtr;
				}
				__syncthreads();
				jacobianMemory[frictionIndex].m_data0[thread] = jacobianInfo.m_data0[thread];
				
				frictionIndex ++;
				__syncthreads();
			}

			if (contact.m_flags & m_friction1Enable__) {

				if (thread < 4) {
					jacobianInfo.m_Jt.m_jacobian_IM0.m_linear[thread] = contact.m_dir1[thread];
					jacobianInfo.m_Jt.m_jacobian_IM0.m_angular[thread] = r0[crossX] * contact.m_dir1[crossY] - r0[crossY] * contact.m_dir1[crossX];			
													
					jacobianInfo.m_Jt.m_jacobian_IM1.m_linear[thread] = -contact.m_dir1[thread];
					jacobianInfo.m_Jt.m_jacobian_IM1.m_angular[thread] = contact.m_dir1[crossX] * r1[crossY] - contact.m_dir1[crossY] * r1[crossX];
				}
				__syncthreads();
				jacobianMemory[frictionIndex].m_Jt.m_jacobianData[thread] = jacobianInfo.m_Jt.m_jacobianData[thread];
				__syncthreads();	
				
				if (thread == 0) {				
					relVelocErr = DotProduct (velocError, contact.m_dir1);
					jacobianInfo.m_normalForceIndex = jacobIndex;
					jacobianInfo.m_diagDamp = 1.0f;
					jacobianInfo.m_restitution = 0.0f;
					jacobianInfo.m_penetration = 0.0f;
					jacobianInfo.m_penetrationStiffness = 0.0f;
					if (contact.m_flags & m_override1Accel__) {
						accel = contact.m_dir1_Force;
						isMotor = 1;
					} else {
						accel = relVelocErr * invTimestep;
						isMotor = 0;
					}
					jacobianInfo.m_coordenateAccel = accel;
					jacobianInfo.m_accelIsMotor = isMotor;
					
					if (fabsf (relVelocErr) > MAX_DYNAMIC_FRICTION_SPEED) {
						lowBound = -contact.m_dynamicFriction1;
						upperBound = contact.m_dynamicFriction1;
					} else {
						lowBound = -contact.m_staticFriction1;
						upperBound = contact.m_staticFriction1;
					}
					
					jacobianInfo.m_lowerBoundFrictionCoefficent = lowBound;
					jacobianInfo.m_upperBoundFrictionCoefficent = upperBound;
					jacobianInfo.m_force = contact.m_dir1_Force;
					jacobianInfo.m_jointFeebackForcePtr = contact.m_dir1_ForcePtr;
				}
				__syncthreads();
				jacobianMemory[frictionIndex].m_data0[thread] = jacobianInfo.m_data0[thread];

				frictionIndex ++;
				__syncthreads();
			}

			__syncthreads();
			jacobIndex ++;
		}

		jointInfo.m_autoPaircount = frictionIndex - jointInfo.m_autoPairstart;

		jointInfo.m_autoPairActiveCount = jointInfo.m_autoPaircount;
		constraintInfo[block].m_data0[thread] = jointInfo.m_data0[thread];
	}
}



__global__ void dgParallelSolverBuildJacobianRows (
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory)
{
	int block;
	int thread;
	int m0;
	int m1;
	int index;
	int count;
	int colum;
	int row;
	float diag;
	float accel;
	float stiffness;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo0;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo1;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianInfo jacobianInfo;	
	__shared__ float tmpMatrix0[16];
	__shared__ float tmpMatrix1[16];
	__shared__ float tmpDiag [4];
	__shared__ float tmpAccel [4];	

	block = blockIdx.x;
	thread = threadIdx.x;

	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];
	colum = thread & 3;
	__syncthreads();

	m0 = jointInfo.m_m0;
	m1 = jointInfo.m_m1;
	count = jointInfo.m_autoPaircount;
	index = jointInfo.m_autoPairstart;

	bodyInfo0.m_invInertiaMatrix[thread] = bodyArray[m0].m_invInertiaMatrix[thread];
	bodyInfo1.m_invInertiaMatrix[thread] = bodyArray[m1].m_invInertiaMatrix[thread];
	
	bodyInfo0.m_data0[thread] = bodyArray[m0].m_data0[thread];
	bodyInfo1.m_data0[thread] = bodyArray[m1].m_data0[thread];
	bodyInfo0.m_data1[thread] = bodyArray[m0].m_data1[thread];
	bodyInfo1.m_data1[thread] = bodyArray[m1].m_data1[thread];	
	__syncthreads();

	for (int i = 0; i < count; i ++) {
		jacobianInfo.m_Jt.m_jacobianData[thread] = jacobianMemory[index].m_Jt.m_jacobianData[thread];
		jacobianInfo.m_data0[thread] = jacobianMemory[index].m_data0[thread];
		__syncthreads();

		tmpMatrix0[thread] = jacobianInfo.m_Jt.m_jacobian_IM0.m_angular[colum] * bodyInfo0.m_invInertiaMatrix[thread];
		tmpMatrix1[thread] = jacobianInfo.m_Jt.m_jacobian_IM1.m_angular[colum] * bodyInfo1.m_invInertiaMatrix[thread];
		__syncthreads();

		if (thread < 4) {
			row = thread << 2;
//			JMinv[index].m_jacobian_IM0.m_angular = invInertia0.UnrotateVector (Jt[index].m_jacobian_IM0.m_angular);
//			JMinv[index].m_jacobian_IM1.m_angular = invInertia1.UnrotateVector (Jt[index].m_jacobian_IM1.m_angular);
			jacobianInfo.m_JMinv.m_jacobian_IM0.m_angular[thread] = tmpMatrix0[row] + tmpMatrix0[row + 1] + tmpMatrix0[row + 2];		
			jacobianInfo.m_JMinv.m_jacobian_IM1.m_angular[thread] = tmpMatrix1[row] + tmpMatrix1[row + 1] + tmpMatrix1[row + 2];					

//			JMinv[index].m_jacobian_IM0.m_linear = Jt[index].m_jacobian_IM0.m_linear.Scale (invMass0);
//			JMinv[index].m_jacobian_IM1.m_linear = Jt[index].m_jacobian_IM1.m_linear.Scale (invMass1);
			jacobianInfo.m_JMinv.m_jacobian_IM0.m_linear[thread] = jacobianInfo.m_Jt.m_jacobian_IM0.m_linear[thread] * bodyInfo0.m_invInertia[3];		
			jacobianInfo.m_JMinv.m_jacobian_IM1.m_linear[thread] = jacobianInfo.m_Jt.m_jacobian_IM1.m_linear[thread] * bodyInfo1.m_invInertia[3];		
		}
		__syncthreads();
		jacobianMemory[index].m_JMinv.m_jacobianData[thread] = jacobianInfo.m_JMinv.m_jacobianData[thread];
		__syncthreads();		

		if (thread < 4) {			
	
//			dgVector tmpDiag (JMinv[index].m_jacobian_IM0.m_linear.CompProduct(Jt[index].m_jacobian_IM0.m_linear));
//			tmpDiag += JMinv[index].m_jacobian_IM0.m_angular.CompProduct(Jt[index].m_jacobian_IM0.m_angular);
//			tmpDiag += JMinv[index].m_jacobian_IM1.m_linear.CompProduct(Jt[index].m_jacobian_IM1.m_linear);
//			tmpDiag += JMinv[index].m_jacobian_IM1.m_angular.CompProduct(Jt[index].m_jacobian_IM1.m_angular);
			
			diag = jacobianInfo.m_JMinv.m_jacobian_IM0.m_linear[thread] * jacobianInfo.m_Jt.m_jacobian_IM0.m_linear[thread] + 
				   jacobianInfo.m_JMinv.m_jacobian_IM0.m_angular[thread] * jacobianInfo.m_Jt.m_jacobian_IM0.m_angular[thread] + 
				   jacobianInfo.m_JMinv.m_jacobian_IM1.m_linear[thread] * jacobianInfo.m_Jt.m_jacobian_IM1.m_linear[thread] + 
				   jacobianInfo.m_JMinv.m_jacobian_IM1.m_angular[thread] * jacobianInfo.m_Jt.m_jacobian_IM1.m_angular[thread]; 
				   
//			dgVector tmpAccel (JMinv[index].m_jacobian_IM0.m_linear.CompProduct(body0->m_accel));
//			tmpAccel += JMinv[index].m_jacobian_IM0.m_angular.CompProduct(body0->m_alpha);
//			tmpAccel += JMinv[index].m_jacobian_IM1.m_linear.CompProduct(body1->m_accel);
//			tmpAccel += JMinv[index].m_jacobian_IM1.m_angular.CompProduct(body1->m_alpha);
			accel = jacobianInfo.m_JMinv.m_jacobian_IM0.m_linear[thread] * bodyInfo0.m_accel[thread] + 
					jacobianInfo.m_JMinv.m_jacobian_IM0.m_angular[thread] * bodyInfo0.m_alpha[thread] +
					jacobianInfo.m_JMinv.m_jacobian_IM1.m_linear[thread] * bodyInfo1.m_accel[thread] + 
					jacobianInfo.m_JMinv.m_jacobian_IM1.m_angular[thread] * bodyInfo1.m_alpha[thread];
					
			tmpDiag[thread] = diag;
			tmpAccel[thread] = accel;	
		}
		__syncthreads();
	

		if (thread == 0) {
			float externalAcceleration;
			externalAcceleration = -(tmpAccel[0] + tmpAccel[1] + tmpAccel[2]);
			
//			m_deltaAccel[index] = externalAcceleration;
			jacobianInfo.m_deltaAccel = externalAcceleration;
			
//			coordenateAccel[index] += extenalAcceleration;
			jacobianInfo.m_coordenateAccel += externalAcceleration;
			
//			force[index] = jointForceFeeback[index][0];
//			jacobianInfo.m_force = jacobianInfo.m_forceValue;

//			_ASSERTE (diagDamp[index] >= float(0.1f));
//			_ASSERTE (diagDamp[index] <= float(100.0f));
//			stiffness = DG_PSD_DAMP_TOL * diagDamp[index];
			stiffness = DG_PSD_DAMP_TOL * jacobianInfo.m_diagDamp;

//			diag = (tmpDiag.m_x + tmpDiag.m_y + tmpDiag.m_z);
			diag = tmpDiag[0] + tmpDiag[1] + tmpDiag[2];
			
//			diagDamp[index] = diag * stiffness;
			jacobianInfo.m_diagDamp = diag * stiffness;

			diag *= (float(1.0f) + stiffness);

//			invDJMinvJt[index] = float(1.0f) / diag;
			jacobianInfo.m_invDJMinvJt = float(1.0f) / diag;
		}
		__syncthreads();
		jacobianMemory[index].m_data0[thread] = jacobianInfo.m_data0[thread];
		
		index ++;
	}
}






__global__ void CalculateRowsAcceleration (
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory,
	float timestep,
	float invTimestep,
	float firstPassCoef)
{
	int block;
	int thread;
	int m0;
	int m1;
	int count;
	int first;
	float vRel;
	float aRel;
	float tmp;
	float restitution;
	float penetrationVeloc;
	float penetrationCorrection;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianPair veloc0;
	__shared__ dgCudaJacobianMemory::dgJacobianPair veloc1;
	__shared__ dgCudaJacobianMemory::dgJacobianPair tmpVeloc;
	__shared__ dgCudaJacobianMemory::dgJacobianInfo jacobianRow;
	
	block = blockIdx.x;
	thread = threadIdx.x;
	
	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];	
	__syncthreads();

	m0 = jointInfo.m_m0;
	m1 = jointInfo.m_m1;
	count = jointInfo.m_autoPaircount;
	first = jointInfo.m_autoPairstart;

//	const dgJacobianPair* const Jt = params.m_Jt;
//	const dgVector& bodyVeloc0 = m_body0->m_veloc;
//	const dgVector& bodyOmega0 = m_body0->m_omega;
//	const dgVector& bodyVeloc1 = m_body1->m_veloc;
//	const dgVector& bodyOmega1 = m_body1->m_omega;

	veloc0.m_jacobianData[thread] = bodyArray[m0].m_data0[thread];
	veloc1.m_jacobianData[thread] = bodyArray[m1].m_data0[thread];
	__syncthreads();
	for (dgInt32 k = 0; k < count; k ++) {
		jacobianRow.m_data0[thread] = jacobianMemory[first].m_data0[thread];
		__syncthreads();
		
		if (!jacobianRow.m_accelIsMotor) {
			jacobianRow.m_Jt.m_jacobianData[thread] = jacobianMemory[first].m_Jt.m_jacobianData[thread];
			__syncthreads();

			if (thread < 8) {			
	//			dgVector relVeloc (Jt[k].m_jacobian_IM0.m_linear.CompProduct(bodyVeloc0));
	//			relVeloc += Jt[k].m_jacobian_IM0.m_angular.CompProduct(bodyOmega0);
				tmpVeloc.m_jacobianData[thread] = jacobianRow.m_Jt.m_jacobian_IM0.m_data[thread] * veloc0.m_jacobianData[thread];

	//			relVeloc += Jt[k].m_jacobian_IM1.m_linear.CompProduct(bodyVeloc1);
	//			relVeloc += Jt[k].m_jacobian_IM1.m_angular.CompProduct(bodyOmega1);
				tmpVeloc.m_jacobianData[thread] += jacobianRow.m_Jt.m_jacobian_IM1.m_data[thread] * veloc1.m_jacobianData[thread];
			}
			__syncthreads();
			
			if (thread < 4) {		
				tmpVeloc.m_jacobian_IM0.m_linear[thread] += tmpVeloc.m_jacobian_IM0.m_angular[thread]; 	
			}
			__syncthreads();
			
			if (thread == 0) {		
//				vRel = relVeloc.m_x + relVeloc.m_y + relVeloc.m_z;
//				aRel = params.m_externAccelaration[k];
				vRel = tmpVeloc.m_jacobian_IM0.m_linear[0] + tmpVeloc.m_jacobian_IM0.m_linear[1] + tmpVeloc.m_jacobian_IM0.m_linear[2];
				aRel = jacobianRow.m_deltaAccel;
							
				if (jacobianRow.m_normalForceIndex < 0) {
//					restitution = dgFloat32 (1.0f);
//					if (vRel <= dgFloat32 (0.0f)) {
//						restitution += params.m_restitution[k];
//					}
					restitution = (vRel <= 0.0f) ? 1.0f + jacobianRow.m_restitution : 1.0f;
					penetrationVeloc = 0.0f;
					
					if (jacobianRow.m_penetration > 1.0e-2f) {
						if (vRel > 0.0f) {
							penetrationCorrection = vRel * timestep;
//							params.m_penetration[k] = GetMax (dgFloat32 (0.0f), params.m_penetration[k] - penetrationCorrection);
							tmp = jacobianRow.m_penetration - penetrationCorrection;
							jacobianRow.m_penetration = (tmp > 0.0f) ? tmp : 0.0f;
						}
//						penetrationVeloc = -(params.m_penetration * params.m_penetrationStiffness[k]);
						penetrationVeloc = - (jacobianRow.m_penetration * jacobianRow.m_penetrationStiffness);
					}

					vRel *= restitution;
//					vRel = GetMin (dgFloat32 (4.0f), vRel + penetrationVeloc);
					tmp = vRel + penetrationVeloc;
					vRel = (tmp < 4.0f) ? tmp : 4.0f;
				}
//				params.m_coordenateAccel[k] =  (aRel - vRel * params.m_invTimeStep);
				jacobianRow.m_coordenateAccel = (aRel - vRel * invTimestep);
			}
			__syncthreads();			
			
			jacobianMemory[first].m_data0[thread] = jacobianRow.m_data0[thread];
			__syncthreads();			
		}
		first ++;
		__syncthreads();
	}
}




__global__ void UpdateBodiesVelocities (
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaBody::dgBodyWorkData* const bodyForceVeloc,
	float timestep)
{
	int row;
	int block;
	int thread;
	__shared__ float tmp[16];
	__shared__ dgCudaJacobian accelAlpha;
	__shared__ dgCudaJacobian forceAlpha;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo;
	__shared__ dgCudaBody::dgBodyWorkData internalForceVeloc;

	block = blockIdx.x;
	thread = threadIdx.x;
	row = thread >> 2;

	bodyInfo.m_data1[thread] = bodyArray[block].m_data1[thread];
	__syncthreads();
	
	if (bodyInfo.m_invInertia[3] > 0.0f) {
		bodyInfo.m_invInertiaMatrix[thread] = bodyArray[block].m_invInertiaMatrix[thread];
		bodyInfo.m_data0[thread] = bodyArray[block].m_data0[thread];	
		internalForceVeloc.m_data0[thread] = bodyForceVeloc[block].m_data0[thread];
		__syncthreads();

	//	dgBody* body;
	//	body = m_bodyArray[i].m_body;
	//	dgVector force (body->m_accel + m_internalForces[i].m_linear);
	//	dgVector torque (body->m_alpha + m_internalForces[i].m_angular);
		if (thread < 8) {
			forceAlpha.m_data[thread] = bodyInfo.m_forceTorque.m_data[thread] + internalForceVeloc.m_internalForce.m_data[thread];
		}
		__syncthreads();	
	
//		dgVector accel (force.Scale (body->m_invMass.m_w));
//		dgVector alpha (body->m_invWorldInertiaMatrix.RotateVector (torque));
		tmp[thread] = forceAlpha.m_angular[row] * bodyInfo.m_invInertiaMatrix[thread];
		__syncthreads();
		if (thread < 4) {
			accelAlpha.m_linear[thread] = forceAlpha.m_linear[thread] * bodyInfo.m_invInertia[3];
			accelAlpha.m_angular[thread] = tmp[thread] + tmp[thread + 4] + tmp[thread + 8];		
		}
		__syncthreads();
		
//		body->m_veloc += accel.Scale(m_timeStep);
//		body->m_omega += alpha.Scale(m_timeStep);
//		m_internalVeloc[i].m_linear += body->m_veloc;
//		m_internalVeloc[i].m_angular += body->m_omega;
		if (thread < 8) {
			bodyInfo.m_veloOmega.m_data[thread] += accelAlpha.m_data[thread] * timestep;
			internalForceVeloc.m_internalVeloc.m_data[thread] += bodyInfo.m_veloOmega.m_data[thread];
		}
		__syncthreads();
		
		bodyArray[block].m_data0[thread] = bodyInfo.m_data0[thread];
		bodyForceVeloc[block].m_data0[thread] = internalForceVeloc.m_data0[thread];
	}
}




__global__ void InitInternalForces (
	int* const jointIndex,
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaBody::dgBodyWorkData* const bodyInternalForces,
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory)
{
	int block;
	int thread;
	int m0;
	int m1;
	int count;
	int first;
	float force;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianPair internalForce;
	
	block = jointIndex[blockIdx.x];
	thread = threadIdx.x;

	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];
	
	internalForce.m_jacobianData[thread] = 0.0f;
	__syncthreads();

	m0 = jointInfo.m_m0;
	m1 = jointInfo.m_m1;
	count = jointInfo.m_autoPaircount;
	first = jointInfo.m_autoPairstart;
	for (int j = 0; j < count; j ++) { 
		force = jacobianMemory[first].m_force;
		internalForce.m_jacobianData[thread] += jacobianMemory[first].m_Jt.m_jacobianData[thread] * force;
		first ++;
	}
	__syncthreads();


	if (thread < 8) {
		bodyInternalForces[m0].m_internalForce.m_data[thread] += internalForce.m_jacobian_IM0.m_data[thread];
		bodyInternalForces[m1].m_internalForce.m_data[thread] += internalForce.m_jacobian_IM1.m_data[thread];		
	}
}

/*
__global__ void InertiaMatricesAndKineticForces (dgCudaBody::dgBodyInfo* const bodies)
{
	int row0;
	int col0;
	int col1;
	int col2;
	int col3;
	int block;
	int blockBase;
	int thread;
	int row0Scale;
	int col0Scale;
	int col1Scale;
	int col2Scale;
	int col3Scale;
	int vectorEntry;	
	float val;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo[1<<DG_CLUSTER_TREAD_SIZE_BITS];
	__shared__ float tmp [1<<DG_CLUSTER_TREAD_SIZE_BITS][16]; 
	__shared__ float transp [1<<DG_CLUSTER_TREAD_SIZE_BITS][16]; 
	
	block = blockIdx.x << DG_CLUSTER_TREAD_SIZE_BITS;
	thread = threadIdx.x & 15;
	blockBase = threadIdx.x >> 4;
	vectorEntry = threadIdx.x >> 2;
	
	row0 = thread >> 2;
	col0 = thread & 3;
	col1 = (thread + 1) & 3;
	col2 = (thread + 2) & 3;
	col3 = (thread + 3) & 3;
	
	row0Scale = row0 << 2;
	col0Scale = col0 << 2;
	col1Scale = col1 << 2;
	col2Scale = col2 << 2;
	col3Scale = col3 << 2;
	
	bodyInfo[blockBase].m_matrix[thread] = bodies[block + blockBase].m_matrix[thread];
	bodyInfo[blockBase].m_data0[thread] = bodies[block + blockBase].m_data0[thread];
	bodyInfo[blockBase].m_data1[thread] = bodies[block + blockBase].m_data1[thread];
	__syncthreads();

	// calculate the inertia Matrix
	val = (col0 < 3) ? bodyInfo[blockBase].m_invInertia[col0] : 0.0f;
	transp[blockBase][thread] = bodyInfo[blockBase].m_matrix[col0Scale + row0] * val;


//	MatrixMultiply(accMatrix, transp, tmpMatrix);	
	bodyInfo[blockBase].m_invInertiaMatrix[thread] = transp[blockBase][row0Scale + col0] * bodyInfo[blockBase].m_matrix[col0Scale + col0] +
													 transp[blockBase][row0Scale + col1] * bodyInfo[blockBase].m_matrix[col1Scale + col0] +
										             transp[blockBase][row0Scale + col2] * bodyInfo[blockBase].m_matrix[col2Scale + col0] +
										             transp[blockBase][row0Scale + col3] * bodyInfo[blockBase].m_matrix[col3Scale + col0];


//	dgVector omega (m_matrix.UnrotateVector (m_omega));
//	MatrixUnrotateVector(tmpOmega, tmpOmega, tmpMatrix);
	tmp[blockBase][thread] = bodyInfo[blockBase].m_omega[col0] * bodyInfo[blockBase].m_matrix[thread];
	__syncthreads();		

	if (threadIdx.x < (4 << DG_CLUSTER_TREAD_SIZE_BITS)) {
//		bodyInfo.m_omega[thread] = tmp[col0Scale] + tmp[col0Scale+1] + tmp[col0Scale+2];		
		val = tmp[vectorEntry][col0Scale] + tmp[vectorEntry][col0Scale + 1] + tmp[vectorEntry][col0Scale + 2];		
		
		//m_veloc -= m_veloc.Scale (m_dampCoef.m_w);
		bodyInfo[vectorEntry].m_veloc[col0] -= bodyInfo[vectorEntry].m_veloc[col0] * bodyInfo[vectorEntry].m_dampCoef[3];
		
		//omega -= omega.CompProduct (m_dampCoef);
//		bodyInfo.m_omega[thread] -= bodyInfo.m_omega[thread] * bodyInfo.m_dampCoef[col0];
		bodyInfo[vectorEntry].m_omega[col0] = val - val * bodyInfo[vectorEntry].m_dampCoef[col0];
	}
	__syncthreads();
	
//	m_omega = m_matrix.RotateVector (omega);
//	MatrixRotateVector(tmpOmega, tmpOmega, tmpMatrix);
	tmp[blockBase][thread] = bodyInfo[blockBase].m_omega[row0] * bodyInfo[blockBase].m_matrix[thread];
	__syncthreads();
	if (threadIdx.x < (4 << DG_CLUSTER_TREAD_SIZE_BITS)) {
		val = tmp[vectorEntry][col0] + tmp[vectorEntry][col0 + 4] + tmp[vectorEntry][col0 + 8];		
		
		bodyInfo[vectorEntry].m_omega[thread] = val;		
		bodyInfo[vectorEntry].m_netForce.m_angular[col0] = val;
		bodyInfo[vectorEntry].m_netForce.m_linear[col0] = bodyInfo[vectorEntry].m_veloc[col0];
	}
	__syncthreads();

	bodies[block + blockBase].m_data0[thread] = bodyInfo[blockBase].m_data0[thread];
	bodies[block + blockBase].m_data2[thread] = bodyInfo[blockBase].m_data2[thread];
	bodies[block + blockBase].m_invInertiaMatrix[thread] = bodyInfo[blockBase].m_invInertiaMatrix[thread];
}
*/

__global__ void InertiaMatricesAndKineticForces (dgCudaBody::dgBodyInfo* const bodies)
{
	int row;
	int colum;
	int block;
	int thread;
	int rowScale;
	int columScale;
	float val;
	__shared__ dgCudaBody::dgBodyInfo bodyInfo;
	__shared__ float tmp [16]; 
	__shared__ float transp [16]; 
	
	block = blockIdx.x;
	thread = threadIdx.x;
	
	row = thread >> 2;
	colum = thread & 3;
	rowScale = row << 2;
	columScale = thread << 2;

	bodyInfo.m_matrix[thread] = bodies[block].m_matrix[thread];
	bodyInfo.m_data0[thread] = bodies[block].m_data0[thread];
	bodyInfo.m_data1[thread] = bodies[block].m_data1[thread];
	__syncthreads();

	// calculate the inertia Matrix
	transp[rowScale + colum] = bodyInfo.m_matrix[(colum << 2) + row] * bodyInfo.m_invInertia[colum];
	__syncthreads();	

//	MatrixMultiply(accMatrix, transp, tmpMatrix);	
	bodyInfo.m_invInertiaMatrix[rowScale + colum] = transp[rowScale + 0] * bodyInfo.m_matrix[0 + colum] +
												    transp[rowScale + 1] * bodyInfo.m_matrix[4 + colum] +
												    transp[rowScale + 2] * bodyInfo.m_matrix[8 + colum];
	
//	dgVector omega (m_matrix.UnrotateVector (m_omega));
//	MatrixUnrotateVector(tmpOmega, tmpOmega, tmpMatrix);
	tmp[thread] = bodyInfo.m_omega[colum] * bodyInfo.m_matrix[thread];
	__syncthreads();		

	if (thread < 4) {
		bodyInfo.m_omega[thread] = tmp[columScale] + tmp[columScale+1] + tmp[columScale+2];		
		
		//m_veloc -= m_veloc.Scale (m_dampCoef.m_w);
		bodyInfo.m_veloc[thread] -= bodyInfo.m_veloc[thread] * bodyInfo.m_dampCoef[3];
	}
	__syncthreads();			
	
	if (thread < 4) {
		//omega -= omega.CompProduct (m_dampCoef);
		bodyInfo.m_omega[thread] -= bodyInfo.m_omega[thread] * bodyInfo.m_dampCoef[thread];
	}
	__syncthreads();
	
//	m_omega = m_matrix.RotateVector (omega);
//	MatrixRotateVector(tmpOmega, tmpOmega, tmpMatrix);
	tmp[thread] = bodyInfo.m_omega[row] * bodyInfo.m_matrix[thread];
	__syncthreads();
	if (thread < 4) {
		val = tmp[thread] + tmp[thread + 4] + tmp[thread + 8];		
		
		bodyInfo.m_omega[thread] = val;		
		bodyInfo.m_netForce.m_angular[thread] = val;
		bodyInfo.m_netForce.m_linear[thread] = bodyInfo.m_veloc[thread];
	}
	__syncthreads();

	bodies[block].m_data0[thread] = bodyInfo.m_data0[thread];
	bodies[block].m_data2[thread] = bodyInfo.m_data2[thread];
	bodies[block].m_invInertiaMatrix[thread] = bodyInfo.m_invInertiaMatrix[thread];
}





#if 0
__global__ void CalculateJointForces (
	int* const jointIndex,
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaBody::dgBodyWorkData* const bodyForceVeloc,
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory)
{
	int m0;
	int m1;
	int count;
	int first;
	int block;
	int thread;
	int frictionIndex;
	float a;
	float f;
	float frictionNormal;
	float lowerFrictionForce;
	float upperFrictionForce;
	__shared__ float prevValue;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianPair rowAccel;	
	__shared__ dgCudaJacobianMemory::dgJacobianInfo jacobianRow;	
	__shared__ dgCudaJacobianMemory::dgJacobianPair internalForce;	

//	block = blockIdx.x;
	block = jointIndex[blockIdx.x];
	thread = threadIdx.x;
	
	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];	
	__syncthreads();


//	currJoint = threadIndex + i;
//	index = m_constraintArray[currJoint].m_autoPairstart;
//	rowsCount = m_constraintArray[currJoint].m_autoPaircount;
//	m0 = m_constraintArray[currJoint].m_m0;
//	m1 = m_constraintArray[currJoint].m_m1;

	m0 = jointInfo.m_m0;
	m1 = jointInfo.m_m1;
	count = jointInfo.m_autoPaircount;
	first = jointInfo.m_autoPairstart;


//	dgVector linearM0 (m_internalForces[m0].m_linear);
//	dgVector angularM0 (m_internalForces[m0].m_angular);
//	dgVector linearM1 (m_internalForces[m1].m_linear);
//	dgVector angularM1 (m_internalForces[m1].m_angular);
	if (thread < 8) {
		internalForce.m_jacobian_IM0.m_data[thread] =  bodyForceVeloc[m0].m_internalForce.m_data[thread];
		internalForce.m_jacobian_IM1.m_data[thread] =  bodyForceVeloc[m1].m_internalForce.m_data[thread];
	}
	prevValue = 0.0f;
	__syncthreads();	
	
	for (int k = 0; k < count; k ++) {
//		dgVector acc (m_JMinv[index].m_jacobian_IM0.m_linear.CompProduct(linearM0));
//		acc += m_JMinv[index].m_jacobian_IM0.m_angular.CompProduct (angularM0);
//		acc += m_JMinv[index].m_jacobian_IM1.m_linear.CompProduct (linearM1);
//		acc += m_JMinv[index].m_jacobian_IM1.m_angular.CompProduct (angularM1);
		rowAccel.m_jacobianData[thread] = jacobianMemory[first].m_JMinv.m_jacobianData[thread] * internalForce.m_jacobianData[thread];
		jacobianRow.m_data0[thread] = jacobianMemory[first].m_data0[thread];		
		__syncthreads();	

		for (int n = 8; n; n >>= 1) {		
			if (thread < n) {	
				rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[thread + n];
			}
			__syncthreads();			
		}

		if (thread == 0) {
//			a = m_coordenateAccel[index] - acc.m_x - acc.m_y - acc.m_z - m_force[index] * m_diagDamp[index];
			a = jacobianRow.m_coordenateAccel - rowAccel.m_jacobianData[0] - jacobianRow.m_force * jacobianRow.m_diagDamp;
			
//			f = m_force[index] + m_invDJMinvJt[index] * a;
			f = jacobianRow.m_force + jacobianRow.m_invDJMinvJt * a;

//			frictionIndex = m_normalForceIndex[index];
//			_ASSERTE (((frictionIndex < 0) && (m_force[frictionIndex] == dgFloat32 (1.0f))) || ((frictionIndex >= 0) && (m_force[frictionIndex] >= dgFloat32 (0.0f))));
//			frictionNormal = m_force[frictionIndex];
			
			frictionNormal = 1.0f;
			frictionIndex = jacobianRow.m_normalForceIndex;
			if (frictionIndex >= 0) {
				frictionNormal = jacobianMemory[frictionIndex].m_force;
			} 

//			lowerFrictionForce = frictionNormal * m_lowerFrictionCoef[index];
//			upperFrictionForce = frictionNormal * m_upperFrictionCoef[index];
			lowerFrictionForce = frictionNormal * jacobianRow.m_lowerBoundFrictionCoefficent;
			upperFrictionForce = frictionNormal * jacobianRow.m_upperBoundFrictionCoefficent;

			if (f > upperFrictionForce) {
				a = dgFloat32 (0.0f);
				f = upperFrictionForce;
			} else if (f < lowerFrictionForce) {
				a = dgFloat32 (0.0f);
				f = lowerFrictionForce;
			}

//			accNorm = GetMax (accNorm, dgAbsf (a));
			prevValue = f - jacobianRow.m_force;
//			jacobianRow.m_force = f;
			jacobianMemory[first].m_force = f;
		}
		__syncthreads();	

//		linearM0 += m_Jt[index].m_jacobian_IM0.m_linear.Scale (prevValue);
//		angularM0 += m_Jt[index].m_jacobian_IM0.m_angular.Scale (prevValue);
//		linearM1 += m_Jt[index].m_jacobian_IM1.m_linear.Scale (prevValue);
//		angularM1 += m_Jt[index].m_jacobian_IM1.m_angular.Scale (prevValue);
		internalForce.m_jacobianData[thread] += jacobianMemory[first].m_Jt.m_jacobianData[thread] * prevValue;
		__syncthreads();	
		first ++;
	}


//	m_internalForces[m0].m_linear = linearM0;
//	m_internalForces[m0].m_angular = angularM0;
//	m_internalForces[m1].m_linear = linearM1;
//	m_internalForces[m1].m_angular = angularM1;
	if (thread < 8) {
		bodyForceVeloc[m0].m_internalForce.m_data[thread] = internalForce.m_jacobian_IM0.m_data[thread];  
		bodyForceVeloc[m1].m_internalForce.m_data[thread] = internalForce.m_jacobian_IM1.m_data[thread];
	}
}

#else

__global__ void CalculateJointForces (
	int* const jointIndex,
	dgCudaJointGraph::dgJointInfo* const constraintInfo,
	dgCudaBody::dgBodyInfo* const bodyArray,
	dgCudaBody::dgBodyWorkData* const bodyForceVeloc,
	dgCudaJacobianMemory::dgJacobianInfo* const jacobianMemory)
{
	int m0;
	int m1;
	int count;
	int first;
	int block;
	int thread;
	int frictionIndex;
	float a;
	float f;
	float frictionNormal;
	float lowerFrictionForce;
	float upperFrictionForce;
	__shared__ float prevValue;
	__shared__ dgCudaJointGraph::dgJointInfo jointInfo;
	__shared__ dgCudaJacobianMemory::dgJacobianPair rowAccel;	
	__shared__ dgCudaJacobianMemory::dgJacobianInfo jacobianRow;	
	__shared__ dgCudaJacobianMemory::dgJacobianPair internalForce;	

//	block = blockIdx.x;
	block = jointIndex[blockIdx.x];
	thread = threadIdx.x;
	
	jointInfo.m_data0[thread] = constraintInfo[block].m_data0[thread];	
	__syncthreads();

	m0 = jointInfo.m_m0;
	m1 = jointInfo.m_m1;
	count = jointInfo.m_autoPaircount;
	first = jointInfo.m_autoPairstart;

	if (thread < 8) {
		internalForce.m_jacobian_IM0.m_data[thread] =  bodyForceVeloc[m0].m_internalForce.m_data[thread];
		internalForce.m_jacobian_IM1.m_data[thread] =  bodyForceVeloc[m1].m_internalForce.m_data[thread];
	}
	prevValue = 0.0f;
	__syncthreads();	
	
	for (int k = 0; k < count; k ++) {
		rowAccel.m_jacobianData[thread] = jacobianMemory[first].m_JMinv.m_jacobianData[thread] * internalForce.m_jacobianData[thread];
		jacobianRow.m_data0[thread] = jacobianMemory[first].m_data0[thread];		
		__syncthreads();	

//		for (int n = 8; n; n >>= 1) {		
//			rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[(thread + n) & 15];
//		}
		rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[(thread + 8) & 15];
		rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[(thread + 4) & 15];
		rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[(thread + 2) & 15];
		rowAccel.m_jacobianData[thread] += rowAccel.m_jacobianData[(thread + 1) & 15];

		a = jacobianRow.m_coordenateAccel - rowAccel.m_jacobianData[0] - jacobianRow.m_force * jacobianRow.m_diagDamp;
		f = jacobianRow.m_force + jacobianRow.m_invDJMinvJt * a;
		frictionIndex = jacobianRow.m_normalForceIndex;
		frictionNormal = (frictionIndex >= 0) ? jacobianMemory[frictionIndex].m_force : 1.0f;
		lowerFrictionForce = frictionNormal * jacobianRow.m_lowerBoundFrictionCoefficent;
		upperFrictionForce = frictionNormal * jacobianRow.m_upperBoundFrictionCoefficent;
		if (f > upperFrictionForce) {
			f = upperFrictionForce;
		} else if (f < lowerFrictionForce) {
			f = lowerFrictionForce;
		}

		prevValue = f - jacobianRow.m_force;
		jacobianMemory[first].m_force = f;
//		__syncthreads();	

		internalForce.m_jacobianData[thread] += jacobianMemory[first].m_Jt.m_jacobianData[thread] * prevValue;
//		__syncthreads();	
		first ++;
	}

	if (thread < 8) {
		bodyForceVeloc[m0].m_internalForce.m_data[thread] = internalForce.m_jacobian_IM0.m_data[thread];  
		bodyForceVeloc[m1].m_internalForce.m_data[thread] = internalForce.m_jacobian_IM1.m_data[thread];
	}
}


#endif